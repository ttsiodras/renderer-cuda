#include "hip/hip_runtime.h"
/*
 *  renderer - A simple implementation of polygon-based 3D algorithms.
 *  Copyright (C) 2004  Thanassis Tsiodras (ttsiodras@gmail.com)
 *
 *  This program is free software; you can redistribute it and/or modify
 *  it under the terms of the GNU General Public License as published by
 *  the Free Software Foundation; either version 2 of the License, or
 *  (at your option) any later version.
 *
 *  This program is distributed in the hope that it will be useful,
 *  but WITHOUT ANY WARRANTY; without even the implied warranty of
 *  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *  GNU General Public License for more details.
 *
 *  You should have received a copy of the GNU General Public License
 *  along with this program; if not, write to the Free Software
 *  Foundation, Inc., 59 Temple Place, Suite 330, Boston, MA  02111-1307  USA
 */

#ifdef _WIN32
#include <windows.h>
#include <GL/glew.h>
#else
#define GL_GLEXT_PROTOTYPES
#include <GL/gl.h>
#endif
#include <GL/glut.h>

#include <SDL.h>

#include <cfloat>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <hip/hip_vector_types.h>

#include "Types.h"
#include "Base3d.h"
#include "Camera.h"

#include "cudarenderer.h"

/////////////////////////////////
// Raytracing configuration
#define THREADS_PER_BLOCK   64

// What depth to stop reflections and refractions?
#define MAX_RAY_DEPTH	    2

// Ray intersections of a distance <=NUDGE_FACTOR (from the origin) don't count
#define NUDGE_FACTOR     1e-5f

// How much the reflected color contributes to the overall
#define REFLECTIONS_RATE 0.375f

//////////////////////////////
// Enable ambient occlusion?
//#define AMBIENT_OCCLUSION
// How many ambient rays to spawn per ray intersection?
#define AMBIENT_SAMPLES  32
// How close to check for ambient occlusion?
#define AMBIENT_RANGE    0.15f

__constant__ unsigned VERTICES;
__constant__ unsigned TRIANGLES;

// Textures for vertices, triangles and BVH data
// (see CudaRender() below, as well as main() to see the data setup process)
texture<uint1, 1, hipReadModeElementType> g_triIdxListTexture;
texture<float2, 1, hipReadModeElementType> g_pCFBVHlimitsTexture;
texture<uint4, 1, hipReadModeElementType> g_pCFBVHindexesOrTrilistsTexture;
texture<float4, 1, hipReadModeElementType> g_verticesTexture;
texture<float4, 1, hipReadModeElementType> g_trianglesTexture;

// Utility functions

// CUDA dot product
__device__ coord dotCUDA(const Vector3& l, const Vector3& r)
{
    return l._x*r._x +l._y*r._y +l._z*r._z;
}

__device__ coord dotCUDA(const float4& l, const Vector3& r)
{
    return l.x*r._x +l.y*r._y +l.z*r._z;
}

__device__ coord dotCUDA(const Vector3& l, const float4& r)
{
    return l._x*r.x +l._y*r.y +l._z*r.z;
}

// CUDA cross
__device__ Vector3 crossCUDA(const Vector3& l, const Vector3& r)
{
    coord x,y,z;
    const coord &aax=l._x;
    const coord &aay=l._y;
    const coord &aaz=l._z;
    const coord &bbx=r._x;
    const coord &bby=r._y;
    const coord &bbz=r._z;
    x=aay*bbz-bby*aaz;
    y=bbx*aaz-aax*bbz;
    z=aax*bby-aay*bbx;
    return Vector3(x,y,z);
}

// CUDA distance of two points
__device__ coord distanceCUDA(const Vector3& a, const Vector3& b)
{
    coord dx=a._x - b._x;
    coord dy=a._y - b._y;
    coord dz=a._z - b._z;
    return sqrt(dx*dx + dy*dy + dz*dz);
}

// Sometime you just want to compare, so no sqrt is needed
__device__ coord distancesqCUDA(const Vector3& a, const Vector3& b)
{
    coord dx=a._x - b._x;
    coord dy=a._y - b._y;
    coord dz=a._z - b._z;
    return dx*dx + dy*dy + dz*dz;
}

// Matrix3x3 multipled by Vector3
__device__ Vector3 multiplyRightWith(const Matrix3& mv, const Vector3& r)
{
    coord xnew = mv._row1._x*r._x + mv._row1._y*r._y + mv._row1._z*r._z;
    coord ynew = mv._row2._x*r._x + mv._row2._y*r._y + mv._row2._z*r._z;
    coord znew = mv._row3._x*r._x + mv._row3._y*r._y + mv._row3._z*r._z;
    return Vector3(xnew, ynew, znew);
}

// Transform Vector3 to any space, given Matrix3 and origin
__device__ Vector3 inline TransformToSomeSpace(Vector3 point, Matrix3 *mv, Vector3 *origin)
{
    point -= *origin;
    return multiplyRightWith(*mv, point);
}

// After transformation in camera space, project and plot (used for point rendering)
#define CLIPPLANEDISTANCE 0.2f

__device__ void inline ProjectAndPlot(const Vector3& xformed, int *pixels, int defaultColor=0x00FFFFFF )
{
    if (xformed._z>CLIPPLANEDISTANCE) {
	int x = (int)(MAXX/2.f  + FOV * xformed._y/xformed._z);
	int y = (int)(MAXY/2.f - FOV * xformed._x/xformed._z);
	if (y>=0.f && y<(int)MAXY && x>=0.f && x<(int)MAXX)
	    pixels[y*MAXX + x] = defaultColor;
    }
}

////////////////////////////////////////
// Rendering kernel for MODE_POINTS
////////////////////////////////////////

__global__ void CoreLoopVertices(int *pixels, Matrix3 *cudaWorldToCameraSpace, Vector3 *eye)
{
    unsigned idx = blockIdx.x*blockDim.x + threadIdx.x;

    if (idx >= VERTICES)
	return;

    // Simple projection and ploting of a white point per vertex

    // Plot projected coordinates (on screen)
    Vector3 v(tex1Dfetch(g_verticesTexture, 2*idx));
    ProjectAndPlot(
	TransformToSomeSpace(v, cudaWorldToCameraSpace, eye),
	pixels);
}

//////////////////////////////////////////////
// Rendering kernel for MODE_POINTSHIDDEN
//////////////////////////////////////////////

// Create OpenGL BGR value for assignment in PBO buffer
__device__ int getColor(Pixel& p)
{
    return (((unsigned)p._b) << 16) | (((unsigned)p._g) << 8) | (((unsigned)p._r));
}

__global__ void CoreLoopTriangles(int *pixels, Matrix3 *cudaWorldToCameraSpace, Triangle *pTriangles, Vector3 *eye)
{
    unsigned idx = blockIdx.x*blockDim.x + threadIdx.x;

    if (idx >= TRIANGLES)
	return;

    // First check if the triangle is visible from where we stand
    // (closed objects only)

    float4 center = tex1Dfetch(g_trianglesTexture, 5*idx);
    float4 normal = tex1Dfetch(g_trianglesTexture, 5*idx+1);

    Vector3 triToEye = *eye;
    triToEye -= center;
    // Normally we would normalize, but since we just need the sign
    // of the dot product (to determine if it facing us or not)...
    //triToEye.normalize();
    //if (!pTriangles[idx]._twoSided && dotCUDA(triToEye, pTriangles[idx]._normal)<0.f)
    if (center.w == 0.f && dotCUDA(triToEye, normal)<0.f)
	return;

    int color = getColor(pTriangles[idx]._colorf);

    // For each of the 3 vertices of triangle j of object i,
    // transform to camera space, project and plot them
    Vector3 v1(tex1Dfetch(g_verticesTexture, 2*pTriangles[idx]._idx1));
    Vector3 v2(tex1Dfetch(g_verticesTexture, 2*pTriangles[idx]._idx2));
    Vector3 v3(tex1Dfetch(g_verticesTexture, 2*pTriangles[idx]._idx3));
    ProjectAndPlot( TransformToSomeSpace(v1, cudaWorldToCameraSpace, eye), pixels, color);
    ProjectAndPlot( TransformToSomeSpace(v2, cudaWorldToCameraSpace, eye), pixels, color);
    ProjectAndPlot( TransformToSomeSpace(v3, cudaWorldToCameraSpace, eye), pixels, color);
}

///////////////////////////////////////////////
// Raytracing modes
///////////////////////////////////////////////

// Helper function, that checks whether a ray intersects a bbox
__device__ bool RayIntersectsBox(
    const Vector3& originInWorldSpace, const Vector3& rayInWorldSpace, int boxIdx)
{
    // set Tnear = - infinity, Tfar = infinity
    //
    // For each pair of planes P associated with X, Y, and Z do:
    //     (example using X planes)
    //     if direction Xd = 0 then the ray is parallel to the X planes, so
    //         if origin Xo is not between the slabs ( Xo < Xl or Xo > Xh) then
    //             return false
    //     else, if the ray is not parallel to the plane then
    //     begin
    //         compute the intersection distance of the planes
    //         T1 = (Xl - Xo) / Xd
    //         T2 = (Xh - Xo) / Xd
    //         If T1 > T2 swap (T1, T2) /* since T1 intersection with near plane */
    //         If T1 > Tnear set Tnear =T1 /* want largest Tnear */
    //         If T2 < Tfar set Tfar="T2" /* want smallest Tfar */
    //         If Tnear > Tfar box is missed so
    //             return false
    //         If Tfar < 0 box is behind ray
    //             return false
    //     end
    // end of for loop
    //
    // If Box survived all above tests, return true with intersection point Tnear and exit point Tfar.

    coord Tnear, Tfar;
    Tnear = -FLT_MAX;
    Tfar = FLT_MAX;

    float2 limits;

#define CHECK_NEAR_AND_FAR_INTERSECTION(c)							    \
    if (rayInWorldSpace._ ## c == 0.f) {							    \
	if (originInWorldSpace._##c < limits.x) return false;					    \
	if (originInWorldSpace._##c > limits.y) return false;					    \
    } else {											    \
	coord T1 = (limits.x - originInWorldSpace._##c)/rayInWorldSpace._##c;			    \
	coord T2 = (limits.y - originInWorldSpace._##c)/rayInWorldSpace._##c;			    \
	if (T1>T2) { coord tmp=T1; T1=T2; T2=tmp; }						    \
	if (T1 > Tnear) Tnear = T1;								    \
	if (T2 < Tfar)  Tfar = T2;								    \
	if (Tnear > Tfar)									    \
	    return false;									    \
	if (Tfar < 0.f)										    \
	    return false;									    \
    }

    limits = tex1Dfetch(g_pCFBVHlimitsTexture, 3*boxIdx); // box.bottom._x/top._x placed in limits.x/limits.y
    CHECK_NEAR_AND_FAR_INTERSECTION(x)
    limits = tex1Dfetch(g_pCFBVHlimitsTexture, 3*boxIdx+1); // box.bottom._y/top._y placed in limits.x/limits.y
    CHECK_NEAR_AND_FAR_INTERSECTION(y)
    limits = tex1Dfetch(g_pCFBVHlimitsTexture, 3*boxIdx+2); // box.bottom._z/top._z placed in limits.x/limits.y
    CHECK_NEAR_AND_FAR_INTERSECTION(z)

    return true;
}

// Templated ray/triangle intersection function - offers two compile-time options:
//
// The first one is used to discriminate between shadow rays (that stop at the first hit)
// and normal rays, that have to find the closest hit.
//
// The second one enables or disables culling of backfacing triangles, and is...
// (a) enabled for the refraction call (which needs both front and back-faces), but
// (b) disabled for reflections and shadow rays.
//
// C++ compile-time power... all lesser languages bow down... :-)
//
template <bool stopAtfirstRayHit, bool doCulling>
__device__ bool BVH_IntersectTriangles(
    // Inputs
    //Triangle *pTriangles,
    const Vector3& origin, const Vector3& ray, unsigned avoidSelf,
    // outputs
    int& pBestTriIdx,
    //
    // both inputs and outputs!
    //
    // for normal rays:
    //  pointHitInWorldSpace (output)
    //  kXX (outputs) perpendicular distances of intersection point from the 3 triangle edges
    //     (used for PhongNormal calculations)
    //
    // for shadow rays:
    //  pointHitInWorldSpace (input) provides the light position
    Vector3& pointHitInWorldSpace,
    coord& kAB, coord& kBC, coord& kCA)
{
    // in the loop below, maintain the closest triangle and the point where we hit it:
    pBestTriIdx = -1;
    coord bestTriDist;

    // light position passed-in pointHitInWorldSpace (only in shadow mode - i.e. stopAtfirstRayHit=true)
    Vector3& lightPos = pointHitInWorldSpace;

    // Compile-time work (stopAtfirstRayHit is template param)
    if (stopAtfirstRayHit)
	// In shadow ray mode, start from light distance
	bestTriDist = distancesqCUDA(origin, lightPos);
    else
	// In normal mode, start from infinity
	bestTriDist = FLT_MAX;

    int stack[BVH_STACK_SIZE];
    int stackIdx = 0;
    stack[stackIdx++] = 0;
    while(stackIdx) {
	int boxIdx = stack[stackIdx-1];
	//CacheFriendlyBVHNode *pCurrent = &cudaBVHNodes[boxIdx];
	stackIdx--;

	uint4 data = tex1Dfetch(g_pCFBVHindexesOrTrilistsTexture, boxIdx);

	// original, "pure" BVH form...
	//if (!pCurrent->IsLeaf()) {

	// cache-friendly BVH form...
	//if (!(pCurrent->u.leaf._count & 0x80000000)) {

	// textured BVH form...
	if (!(data.x & 0x80000000)) {
	    if (RayIntersectsBox(origin, ray, boxIdx)) {
		//stack[stackIdx++] = pCurrent->u.inner._idxRight;
		stack[stackIdx++] = data.y;
		//stack[stackIdx++] = pCurrent->u.inner._idxLeft;
		stack[stackIdx++] = data.z;
		if(stackIdx>BVH_STACK_SIZE)
		{
		    return false; // XXX
		}
	    }
	} else {
	    // original, "pure" BVH form...
	    //BVHLeaf *p = dynamic_cast<BVHLeaf*>(pCurrent);
	    //for(std::list<const Triangle*>::iterator it=p->_triangles.begin();
	    //    it != p->_triangles.end();
	    //    it++)

	    // cache-friendly BVH form...
	    //for(unsigned i=pCurrent->u.leaf._startIndexInTriIndexList;
	    //    i<pCurrent->u.leaf._startIndexInTriIndexList + (pCurrent->u.leaf._count & 0x7fffffff);

	    // textured BVH form...
	    for(unsigned i=data.w; i<data.w + (data.x & 0x7fffffff); i++) {
		// original, "pure" BVH form...
		//const Triangle& triangle = *(*it);

		// cache-friendly BVH form...
		//const Triangle& triangle = pTriangles[cudaTriIdxList[i]];

		// textured BVH form...
		int idx = tex1Dfetch(g_triIdxListTexture, i).x;

		if (avoidSelf == idx)
		    continue; // avoid self-reflections/refractions

		float4 center = tex1Dfetch(g_trianglesTexture, 5*idx);
		float4 normal = tex1Dfetch(g_trianglesTexture, 5*idx+1);

		// doCulling is a compile-time param, this code will be "codegenerated"
		// at compile time only for reflection-related calls to Raytrace (see below)
		//if (doCulling && !triangle._twoSided) {
		if (doCulling && (center.w == 0.f)) { // template-param, compile-time check
		    // Check visibility of triangle via dot product
		    Vector3 fromTriToOrigin = origin;
		    //fromTriToOrigin -= triangle._center;
		    fromTriToOrigin -= center;
		    // Normally we would normalize, but since we just need the sign
		    // of the dot product (to determine if it facing us or not)...
		    //fromTriToOrigin.normalize();
		    if (dotCUDA(fromTriToOrigin, normal)<0)
			continue;
		}

		// Use the pre-computed triangle intersection data: normal, d, e1/d1, e2/d2, e3/d3
		coord k = dotCUDA(normal, ray);
		if (k == 0.0f)
		    continue; // this triangle is parallel to the ray, ignore it.

		coord s = (normal.w - dotCUDA(normal, origin))/k;
		if (s <= 0.0f) // this triangle is "behind" the origin.
		    continue;
		if (s <= NUDGE_FACTOR)
		    continue;
		Vector3 hit = ray*s;
		hit += origin;

		// Is the intersection of the ray with the triangle's plane INSIDE the triangle?
		float4 ee1 = tex1Dfetch(g_trianglesTexture, 5*idx+2);
		coord kt1 = dotCUDA(ee1, hit) - ee1.w; if (kt1<0.0f) continue;
		float4 ee2 = tex1Dfetch(g_trianglesTexture, 5*idx+3);
		coord kt2 = dotCUDA(ee2, hit) - ee2.w; if (kt2<0.0f) continue;
		float4 ee3 = tex1Dfetch(g_trianglesTexture, 5*idx+4);
		coord kt3 = dotCUDA(ee3, hit) - ee3.w; if (kt3<0.0f) continue;

		// It is, "hit" is the world space coordinate of the intersection.

		// Was this a normal ray or a shadow ray? (template param)
		if (stopAtfirstRayHit) {
		    // Shadow ray, check whether the triangle obstructs the light
		    coord dist = distancesqCUDA(lightPos, hit);
		    if (dist < bestTriDist) // distance to light (squared) passed in kAB
			return true; // we found a triangle obstructing the light, return true
		} else {
		    // Normal ray - it this intersection closer than all the others?
		    coord hitZ = distancesqCUDA(origin, hit);
		    if (hitZ < bestTriDist) {
			// maintain the closest hit
			bestTriDist = hitZ;
			pBestTriIdx = idx;
			pointHitInWorldSpace = hit;
			kAB = kt1;
			kBC = kt2;
			kCA = kt3;
		    }
		}
	    }
	}
    }
    // Normal ray or shadow ray? (compile-time template param)
    if (!stopAtfirstRayHit)
	// for normal ray, return true if we pierced a triangle
	return pBestTriIdx != -1;
    else
	// for shadow ray, return true if we found a triangle obstructing the light.
	return false;
}

// CUDA 1.2 has no recursion - I therefore use the magic of C++ templates:
// Compile-time recursion using the "depth" param!
template <int depth, bool doSpecular, bool doPhongInterp, bool doReflections, bool doShadows, bool doCulling>
__device__ Pixel Raytrace(
    Vector3 originInWorldSpace, Vector3 rayInWorldSpace, int avoidSelf,
    Triangle *pTriangles,
    Vector3 *cudaEyePosInWorldSpace, Vector3 *cudaLightPosInWorldSpace)
{
    int pBestTriIdx = -1;
    const Triangle *pBestTri = NULL;
    Vector3 pointHitInWorldSpace;
    coord kAB=0.f, kBC=0.f, kCA=0.f; // distances from the 3 edges of the triangle (from where we hit it)

    // Use the surface-area heuristic based, bounding volume hierarchy of axis-aligned bounding boxes
    // (keywords: SAH, BVH, AABB)
    if (!BVH_IntersectTriangles<false,doCulling>(
	    //pTriangles,
	    originInWorldSpace, rayInWorldSpace, avoidSelf,
	    pBestTriIdx, pointHitInWorldSpace, kAB, kBC, kCA))
	// We pierced no triangle, return with no contribution (ambient is black)
	return Pixel(0.f,0.f,0.f);

    // Set this to pass to recursive calls below, so that we don't get self-shadow or self-reflection
    // from this triangle...
    avoidSelf = pBestTriIdx;
    pBestTri = &pTriangles[pBestTriIdx];

    // We'll also calculate the color contributed from this intersection
    // Start from the triangle's color
    Pixel color = pBestTri->_colorf;

    // Phong interpolation of normal vector: these values are only set if
    // the doPhongInterp template param is set
    Vector3 phongNormal;
    coord ABx,BCx,CAx,area;
    float4 V1;
    float4 N1;
    float4 V2;
    float4 N2;
    float4 V3;
    float4 N3;
    V1 = tex1Dfetch(g_verticesTexture, 2*pBestTri->_idx1);
    V2 = tex1Dfetch(g_verticesTexture, 2*pBestTri->_idx2);
    V3 = tex1Dfetch(g_verticesTexture, 2*pBestTri->_idx3);
    if (doPhongInterp) { // template-param, compile-time check
	// These are the closest triangle's vertices...
	N1 = tex1Dfetch(g_verticesTexture, 2*pBestTri->_idx1+1);
	N2 = tex1Dfetch(g_verticesTexture, 2*pBestTri->_idx2+1);
	N3 = tex1Dfetch(g_verticesTexture, 2*pBestTri->_idx3+1);
	const Vector3 bestTriA = Vector3(V1.x,V1.y,V1.z);
	const Vector3 bestTriB = Vector3(V2.x,V2.y,V2.z);
	const Vector3 bestTriC = Vector3(V3.x,V3.y,V3.z);
	// ...and their normal vectors:
	const Vector3 bestTriNrmA = Vector3(N1.x,N1.y,N1.z);
	const Vector3 bestTriNrmB = Vector3(N2.x,N2.y,N2.z);
	const Vector3 bestTriNrmC = Vector3(N3.x,N3.y,N3.z);

	// We now want to interpolate the triangle's normal,
	// so that as the "pointHitInWorldSpace" gets closer to
	// a vertex X, the interpolated normal becomes closer to bestTriNrmX,
	// and becomes EXACTLY bestTriNrmX, if the pointHitInWorldSpace is X.
	//
	// To do that, we use the 3 areas of the triangle, as it is divided
	// by the pointHitInWorldSpace.
	//
	// This is the total triangle's area: cross product of two edges
	// (in fact, we should divide by 2, but since we're only interested
	//  in ratios (see below), there is no need)
	Vector3 AB = bestTriB; AB-= bestTriA;  // edge AB
	Vector3 BC = bestTriC; BC-= bestTriB;  // edge BC
	Vector3 crossAB_BC = crossCUDA(AB, BC);
	area = crossAB_BC.length();      // 2*area(ABC)

	// And these are the three sub-triangles - kAB,kBC,kCA were found above...
	ABx = kAB*distanceCUDA(bestTriA, bestTriB);
	BCx = kBC*distanceCUDA(bestTriB, bestTriC);
	CAx = kCA*distanceCUDA(bestTriC, bestTriA);

	// use the area of the sub-triangle ACROSS a point, to scale!
	// (which means that if pointHitInCameraSpace is exactly on a vertex,
	//  the area of the sub-triangle becomes the area of the triangle!)
	Vector3 phongNormalA = bestTriNrmA; phongNormalA *= BCx / area;
	Vector3 phongNormalB = bestTriNrmB; phongNormalB *= CAx / area;
	Vector3 phongNormalC = bestTriNrmC; phongNormalC *= ABx / area;

	// and finally, accumulate the three contributions and normalize.
	phongNormal = phongNormalA + phongNormalB + phongNormalC;
	phongNormal.normalize();
    } else
	phongNormal = pBestTri->_normal;

#ifdef AMBIENT_OCCLUSION
    // Calculate ambient occlusion - throw AMBIENT_SAMPLES number of random rays
    // in the hemisphere formed from the pointHitInWorldSpace and the normal vector...
    int i=0;
    coord totalLight = 0.f, maxLight = 0.f;
    while (i<AMBIENT_SAMPLES) {
	Vector3 ambientRay = phongNormal;
	ambientRay._x += float(rand()-RAND_MAX/2)/(RAND_MAX/2);
	ambientRay._y += float(rand()-RAND_MAX/2)/(RAND_MAX/2);
	ambientRay._z += float(rand()-RAND_MAX/2)/(RAND_MAX/2);
	float cosangle = dotCUDA(ambientRay, phongNormal);
	if (cosangle<0.f) continue;
	i++;
	maxLight += cosangle;
	ambientRay.normalize();
	Vector3 temp(pointHitInWorldSpace);
	temp += ambientRay*AMBIENT_RANGE;
	const Triangle *dummy;
	// Some objects needs a "nudge", to avoid self-shadowing
	//Vector3 nudgedPointHitInWorldSpace = pointHitInWorldSpace;
	//nudgedPointHitInWorldSpace += ambientRay*.005f;
	//if (!BVH_IntersectTriangles<true,true>(
	//	    nudgedPointHitInWorldSpace, ambientRay, avoidSelf,
	if (!BVH_IntersectTriangles<true,true>(
		pTriangles,
		pointHitInWorldSpace, ambientRay, avoidSelf,
		dummy, temp, kAB, kAB, kAB)) {
	    // Accumulate contribution of this random ray
	    totalLight += cosangle;
	}
    }
    // total ambient light, averaged over all random rays
    color *= (AMBIENT/255.0f)*(totalLight/maxLight);
#else
    // Dont calculate ambient occlusion, use the pre-calculated value from the model
    // (assuming it exists!)
    coord ambientOcclusionCoeff;
    if (doPhongInterp) { // template-param, compile-time check
	// we have a phong normal, so use the subtriangle areas
	// to interpolate the 3 ambientOcclusionCoeff values
	ambientOcclusionCoeff =
	    V1.w*BCx/area +
	    V2.w*CAx/area +
	    V3.w*ABx/area;
	    /*pVertices[pBestTri->_idx1]._ambientOcclusionCoeff*BCx/area +
	    pVertices[pBestTri->_idx2]._ambientOcclusionCoeff*CAx/area +
	    pVertices[pBestTri->_idx3]._ambientOcclusionCoeff*ABx/area;*/
    } else {
	// we dont have a phong normal, just average the 3 values of the vertices
	ambientOcclusionCoeff = (V1.w + V2.w + V3.w)/3.f;
	    /*pVertices[pBestTri->_idx1]._ambientOcclusionCoeff +
	    pVertices[pBestTri->_idx2]._ambientOcclusionCoeff +
	    pVertices[pBestTri->_idx3]._ambientOcclusionCoeff)/3.f;*/
    }
    coord ambientFactor = (coord) ((AMBIENT*ambientOcclusionCoeff/255.0f)/255.0f);
    color *= ambientFactor;
#endif // AMBIENT_OCCLUSION

/*
    // Now, for all the lights...
    for(unsigned i=0; i<scene._lights.size(); i++) {
	Light& light = *scene._lights[i];
*/

    Vector3& light = *cudaLightPosInWorldSpace;

    // This light's diffuse and specular contribution
    Pixel dColor = Pixel(); // start with black

    // We calculate the vector from point hit, to light (both in world space).
    Vector3 pointToLight = light;
    pointToLight -= pointHitInWorldSpace;

    bool inShadow = false;

    if (doShadows) { // template-param, compile-time check
	// this is our distance from the light (squared, i.e. we didnt use an sqrt)
	coord distanceFromLightSq = pointToLight.lengthsq();

	Vector3 shadowrayInWorldSpace = pointToLight;
	shadowrayInWorldSpace /= sqrt(distanceFromLightSq);

	int pDummy; // just to fill-in the param, not used for shadowrays
	if (BVH_IntersectTriangles<true,doCulling>(
	    //pTriangles,
	    pointHitInWorldSpace, shadowrayInWorldSpace, avoidSelf,
	    pDummy, // dummy
	    light,
	    kAB, kAB, kAB)) // dummies
	{
	    //continue; // we were in shadow, go to next light
	    inShadow = true;
	}
    }

    if (!inShadow) {
	// Diffuse color
	pointToLight.normalize();  // vector from point to light (in world space)

	coord intensity = dotCUDA(phongNormal, pointToLight);
	if (intensity<0.f) {
	    ; // in shadow, let it be in ambient
	} else {
	    Pixel diffuse = pBestTri->_colorf;
	    diffuse *= (coord) (DIFFUSE*intensity/255.f);   // diffuse set to a maximum of 130/255
	    dColor += diffuse;

	    if (doSpecular) { // template-param, compile-time check
		// Specular color
		// We will use the half vector: pointToLight + point to camera
		Vector3 pointToCamera = *cudaEyePosInWorldSpace;
		pointToCamera -= pointHitInWorldSpace;
		pointToCamera.normalize();

		Vector3 half = pointToLight;
		half += pointToCamera;
		half.normalize();

		// use the interpolated phong normal!
		coord intensity2 = dotCUDA(half, phongNormal);
		if (intensity2>0.f) {
		    intensity2 *= intensity2;
		    intensity2 *= intensity2;
		    intensity2 *= intensity2;
		    intensity2 *= intensity2;
		    intensity2 *= intensity2;
		    dColor += Pixel(
			(unsigned char)(SPECULAR*intensity2),
			(unsigned char)(SPECULAR*intensity2),
			(unsigned char)(SPECULAR*intensity2));
		}
	    }
	}
	color += dColor;
    }

    // } for each light

    if (!doReflections)
	return color;
    else {
	originInWorldSpace = pointHitInWorldSpace;
	const Vector3& nrm = phongNormal;
	float c1 = -dotCUDA(rayInWorldSpace, nrm);

	// Reflections:
	//
	// ray = ray - 2 (ray dot normal) normal
	Vector3 reflectedRay = rayInWorldSpace;
	reflectedRay += nrm*(2.0f*c1);
	reflectedRay.normalize();

	return
	    color
	    /* use backface culling for reflection rays: <true> */
	    + Raytrace<depth+1, doSpecular, doPhongInterp, doReflections, doShadows, true>(
		originInWorldSpace, reflectedRay, avoidSelf,
		pTriangles,
		cudaEyePosInWorldSpace, cudaLightPosInWorldSpace) * REFLECTIONS_RATE
	    /* ...but not for refraction rays: <false>
	      REMOVED, 2011/02/04
	    */
	    ;
    }
}

// CUDA 1.2 has no recursion - I therefore use the magic of C++ templates:
// Compile-time recursion using the "depth" param!
//
// These are the template specializations that stop the compile-time recursion
// at MAX_RAY_DEPTH level.

#define STOP_RECURSION(a,b,c,d,e)							    \
template <>										    \
__device__ Pixel Raytrace<MAX_RAY_DEPTH,a,b,c,d,e>(					    \
    Vector3 originInWorldSpace, Vector3 rayInWorldSpace, int avoidSelf,			    \
    Triangle *pTriangles,								    \
    Vector3 *cudaEyePosInWorldSpace, Vector3 *cudaLightPosInWorldSpace)			    \
{											    \
    return Pixel(0.f,0.f,0.f);								    \
}

#define f false
#define t true
STOP_RECURSION(f,f,f,f,f)
STOP_RECURSION(f,f,f,f,t)
STOP_RECURSION(f,f,f,t,f)
STOP_RECURSION(f,f,f,t,t)
STOP_RECURSION(f,f,t,f,f)
STOP_RECURSION(f,f,t,f,t)
STOP_RECURSION(f,f,t,t,f)
STOP_RECURSION(f,f,t,t,t)
STOP_RECURSION(f,t,f,f,f)
STOP_RECURSION(f,t,f,f,t)
STOP_RECURSION(f,t,f,t,f)
STOP_RECURSION(f,t,f,t,t)
STOP_RECURSION(f,t,t,f,f)
STOP_RECURSION(f,t,t,f,t)
STOP_RECURSION(f,t,t,t,f)
STOP_RECURSION(f,t,t,t,t)
STOP_RECURSION(t,f,f,f,f)
STOP_RECURSION(t,f,f,f,t)
STOP_RECURSION(t,f,f,t,f)
STOP_RECURSION(t,f,f,t,t)
STOP_RECURSION(t,f,t,f,f)
STOP_RECURSION(t,f,t,f,t)
STOP_RECURSION(t,f,t,t,f)
STOP_RECURSION(t,f,t,t,t)
STOP_RECURSION(t,t,f,f,f)
STOP_RECURSION(t,t,f,f,t)
STOP_RECURSION(t,t,f,t,f)
STOP_RECURSION(t,t,f,t,t)
STOP_RECURSION(t,t,t,f,f)
STOP_RECURSION(t,t,t,f,t)
STOP_RECURSION(t,t,t,t,f)
STOP_RECURSION(t,t,t,t,t)
#undef f
#undef t

// Main CUDA kernel, templated, to support each of the desired features:
//
// - using specular lights or not
// - doing Phong normal interpolation or not
// - doing reflections or not
// - doing shadows or not
// - doing anti-alias or not
//
template <bool doSpecular, bool doPhongInterp, bool doReflections, bool doShadows, bool antialias>
__global__ void CoreLoopTrianglesRaycaster(
    int *pixels,
    Matrix3 *cudaWorldToCameraSpace,
    Triangle *pTriangles,
    Vector3 *cudaEyePosInWorldSpace, Vector3 *cudaLightPosInWorldSpace,
    unsigned *cudaMortonTable)
{
    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;
    if (idx>=MAXX*MAXY)
	return;
    // without MORTON_ORDER
    // int x = idx % MAXX;
    // int y = idx / MAXX;

    // This code will not allocate idx to a normal, left-to-right, top-to-bottom
    // allocation of pixels. Instead, it will use the pre-calculated Morton order
    // of pixels in screen space, to make threads end up reading the same (hopefully)
    // BVH and triangle data (more co-alesced, cache-friendly accesses).
    int x = int(cudaMortonTable[idx] & 0xFFFF);
    int y = int((cudaMortonTable[idx] & 0xFFFF0000)>>16);

    Pixel finalColor(0,0,0);
    int pixelsTraced = 1;
    if (antialias)
	pixelsTraced = 4;

    while(pixelsTraced--) {
	// We will shoot a ray in camera space (from Eye to the screen point, so in camera
	// space, from (0,0,0) to this:
	coord xx = (coord)x;
	coord yy = (coord)y;

	if (antialias) {
	    // nudge in a cross pattern around the pixel center
	    xx += 0.25f - .5f*(pixelsTraced&1);
	    yy += 0.25f - .5f*((pixelsTraced&2)>>1);
	}
	coord lx = coord((MAXY/2)-yy)/SCREEN_DIST;
	coord ly = coord(xx-(MAXX/2))/SCREEN_DIST;
	coord lz = 1.0f;
	Vector3 rayInCameraSpace(lx,ly,lz);
	rayInCameraSpace.normalize();

	// We will need the origin in world space
	Vector3 originInWorldSpace = *cudaEyePosInWorldSpace;

	// We have a rayInCameraSpace, and we want to use the BVH, which was constructed
	// in World space, so we convert the ray in World space
	Vector3 rayInWorldSpace = cudaWorldToCameraSpace->_row1 * rayInCameraSpace._x;
	rayInWorldSpace += cudaWorldToCameraSpace->_row2 * rayInCameraSpace._y;
	rayInWorldSpace += cudaWorldToCameraSpace->_row3 * rayInCameraSpace._z;
	// in theory, this should not be required
	rayInWorldSpace.normalize();

	// Primary ray, level 0, and we want backface culling: <true>
	finalColor += Raytrace<0, doSpecular, doPhongInterp, doReflections, doShadows, true>(
	    originInWorldSpace, rayInWorldSpace, -1,
	    pTriangles,
	    cudaEyePosInWorldSpace, cudaLightPosInWorldSpace);
    }
    if (antialias)
	finalColor /= 4.f;
    if (finalColor._r>255.0f) finalColor._r=255.0f;
    if (finalColor._g>255.0f) finalColor._g=255.0f;
    if (finalColor._b>255.0f) finalColor._b=255.0f;

    int color = getColor(finalColor);
    // without MORTON_ORDER
    //pixels[idx] = color;
    pixels[y*MAXX+x] = color;
}

// The bridge to the normal C++ world: templated, to include only the mode-specific code in each incantation

bool g_bFirstTime = true;

void CudaRender(
    Matrix3 *cudaWorldToCameraSpace,
    Vertex *cudaPtrVertices, Triangle *cudaPtrTriangles, float *cudaTriangleIntersectionData,
    int *cudaTriIdxList, float *cudaBVHlimits, int *cudaBVHindexesOrTrilists,
    Vector3 *cudaEyePosInWorldSpace, Vector3 *cudaLightPosInWorldSpace,
    unsigned *cudaMortonTable)
{
    if (g_bFirstTime) {
	// The first time any of the CudaRender templates is called,
	// bind the texture data!
	g_bFirstTime = false;

	hipChannelFormatDesc channel1desc = hipCreateChannelDesc<uint1>();
	hipBindTexture(NULL, &g_triIdxListTexture, cudaTriIdxList, &channel1desc, g_triIndexListNo*sizeof(uint1));

	hipChannelFormatDesc channel2desc = hipCreateChannelDesc<float2>();
	hipBindTexture(NULL, &g_pCFBVHlimitsTexture, cudaBVHlimits, &channel2desc, g_pCFBVH_No*6*sizeof(float));

	hipChannelFormatDesc channel3desc = hipCreateChannelDesc<int4>();
	hipBindTexture(NULL, &g_pCFBVHindexesOrTrilistsTexture, cudaBVHindexesOrTrilists, &channel3desc,
	    g_pCFBVH_No*sizeof(uint4));

	hipChannelFormatDesc channel4desc = hipCreateChannelDesc<float4>();
	hipBindTexture(NULL, &g_verticesTexture, cudaPtrVertices, &channel4desc, g_verticesNo*8*sizeof(float));

	hipChannelFormatDesc channel5desc = hipCreateChannelDesc<float4>();
	hipBindTexture(NULL, &g_trianglesTexture, cudaTriangleIntersectionData, &channel5desc, g_trianglesNo*20*sizeof(float));
    }

    int *pixels;
    glBindTexture(GL_TEXTURE_2D, tex);
    SAFE(cudaGLMapBufferObject((void**)&pixels, buffer));

    if (g_bUsePoints) {
	hipMemset(pixels, 0x40, MAXX*MAXY*sizeof(unsigned)); // Clear all pixels to ambient
	int blocksVertices = (g_verticesNo + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
	CoreLoopVertices<<< blocksVertices, THREADS_PER_BLOCK >>>(
	    pixels, cudaWorldToCameraSpace, cudaEyePosInWorldSpace);
    /*
	hipMemset(pixels, 0x40, MAXX*MAXY*sizeof(unsigned)); // Clear all pixels to ambient
	int blocksTriangles = (g_trianglesNo + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
	CoreLoopTriangles<<< blocksTriangles, THREADS_PER_BLOCK >>>(
	    pixels, cudaWorldToCameraSpace, cudaPtrTriangles, cudaEyePosInWorldSpace);*/
    } else {
	int blockPixels = (MAXY*MAXX + THREADS_PER_BLOCK - 1)/THREADS_PER_BLOCK;
	DBG_PUTS(1, "CoreLoopTrianglesRaycaster begins");

	#define PAINT(bDoSpecular,bDoPhongInterp,bDoReflections,bDoShadows,bDoAntialias)				\
	    CoreLoopTrianglesRaycaster<bDoSpecular,bDoPhongInterp,bDoReflections,bDoShadows,bDoAntialias>		\
	    <<< blockPixels, THREADS_PER_BLOCK >>>(									\
		pixels,													\
		cudaWorldToCameraSpace,											\
		cudaPtrTriangles,											\
		cudaEyePosInWorldSpace, cudaLightPosInWorldSpace,							\
		cudaMortonTable);

	// Being templated, one can use the CoreLoopTrianglesRaycaster in any form one wants.
	// In theory, C++ compilers don't need this expansion (which I wrote with a simple Python script)
	// Unfortunately, we can't pass runtime vars in template params, not even when they are enumerants
	// or booleans...

	if (!g_bUseSpecular && !g_bUsePhongInterp && !g_bUseReflections && !g_bUseShadows && !g_bUseAntialiasing) {
	   PAINT( false , false , false , false , false )
	} else if (!g_bUseSpecular && !g_bUsePhongInterp && !g_bUseReflections && !g_bUseShadows && g_bUseAntialiasing) {
	   PAINT( false , false , false , false , true )
	} else if (!g_bUseSpecular && !g_bUsePhongInterp && !g_bUseReflections && g_bUseShadows && !g_bUseAntialiasing) {
	   PAINT( false , false , false , true , false )
	} else if (!g_bUseSpecular && !g_bUsePhongInterp && !g_bUseReflections && g_bUseShadows && g_bUseAntialiasing) {
	   PAINT( false , false , false , true , true )
	} else if (!g_bUseSpecular && !g_bUsePhongInterp && g_bUseReflections && !g_bUseShadows && !g_bUseAntialiasing) {
	   PAINT( false , false , true , false , false )
	} else if (!g_bUseSpecular && !g_bUsePhongInterp && g_bUseReflections && !g_bUseShadows && g_bUseAntialiasing) {
	   PAINT( false , false , true , false , true )
	} else if (!g_bUseSpecular && !g_bUsePhongInterp && g_bUseReflections && g_bUseShadows && !g_bUseAntialiasing) {
	   PAINT( false , false , true , true , false )
	} else if (!g_bUseSpecular && !g_bUsePhongInterp && g_bUseReflections && g_bUseShadows && g_bUseAntialiasing) {
	   PAINT( false , false , true , true , true )
	} else if (!g_bUseSpecular && g_bUsePhongInterp && !g_bUseReflections && !g_bUseShadows && !g_bUseAntialiasing) {
	   PAINT( false , true , false , false , false )
	} else if (!g_bUseSpecular && g_bUsePhongInterp && !g_bUseReflections && !g_bUseShadows && g_bUseAntialiasing) {
	   PAINT( false , true , false , false , true )
	} else if (!g_bUseSpecular && g_bUsePhongInterp && !g_bUseReflections && g_bUseShadows && !g_bUseAntialiasing) {
	   PAINT( false , true , false , true , false )
	} else if (!g_bUseSpecular && g_bUsePhongInterp && !g_bUseReflections && g_bUseShadows && g_bUseAntialiasing) {
	   PAINT( false , true , false , true , true )
	} else if (!g_bUseSpecular && g_bUsePhongInterp && g_bUseReflections && !g_bUseShadows && !g_bUseAntialiasing) {
	   PAINT( false , true , true , false , false )
	} else if (!g_bUseSpecular && g_bUsePhongInterp && g_bUseReflections && !g_bUseShadows && g_bUseAntialiasing) {
	   PAINT( false , true , true , false , true )
	} else if (!g_bUseSpecular && g_bUsePhongInterp && g_bUseReflections && g_bUseShadows && !g_bUseAntialiasing) {
	   PAINT( false , true , true , true , false )
	} else if (!g_bUseSpecular && g_bUsePhongInterp && g_bUseReflections && g_bUseShadows && g_bUseAntialiasing) {
	   PAINT( false , true , true , true , true )
	} else if (g_bUseSpecular && !g_bUsePhongInterp && !g_bUseReflections && !g_bUseShadows && !g_bUseAntialiasing) {
	   PAINT( true , false , false , false , false )
	} else if (g_bUseSpecular && !g_bUsePhongInterp && !g_bUseReflections && !g_bUseShadows && g_bUseAntialiasing) {
	   PAINT( true , false , false , false , true )
	} else if (g_bUseSpecular && !g_bUsePhongInterp && !g_bUseReflections && g_bUseShadows && !g_bUseAntialiasing) {
	   PAINT( true , false , false , true , false )
	} else if (g_bUseSpecular && !g_bUsePhongInterp && !g_bUseReflections && g_bUseShadows && g_bUseAntialiasing) {
	   PAINT( true , false , false , true , true )
	} else if (g_bUseSpecular && !g_bUsePhongInterp && g_bUseReflections && !g_bUseShadows && !g_bUseAntialiasing) {
	   PAINT( true , false , true , false , false )
	} else if (g_bUseSpecular && !g_bUsePhongInterp && g_bUseReflections && !g_bUseShadows && g_bUseAntialiasing) {
	   PAINT( true , false , true , false , true )
	} else if (g_bUseSpecular && !g_bUsePhongInterp && g_bUseReflections && g_bUseShadows && !g_bUseAntialiasing) {
	   PAINT( true , false , true , true , false )
	} else if (g_bUseSpecular && !g_bUsePhongInterp && g_bUseReflections && g_bUseShadows && g_bUseAntialiasing) {
	   PAINT( true , false , true , true , true )
	} else if (g_bUseSpecular && g_bUsePhongInterp && !g_bUseReflections && !g_bUseShadows && !g_bUseAntialiasing) {
	   PAINT( true , true , false , false , false )
	} else if (g_bUseSpecular && g_bUsePhongInterp && !g_bUseReflections && !g_bUseShadows && g_bUseAntialiasing) {
	   PAINT( true , true , false , false , true )
	} else if (g_bUseSpecular && g_bUsePhongInterp && !g_bUseReflections && g_bUseShadows && !g_bUseAntialiasing) {
	   PAINT( true , true , false , true , false )
	} else if (g_bUseSpecular && g_bUsePhongInterp && !g_bUseReflections && g_bUseShadows && g_bUseAntialiasing) {
	   PAINT( true , true , false , true , true )
	} else if (g_bUseSpecular && g_bUsePhongInterp && g_bUseReflections && !g_bUseShadows && !g_bUseAntialiasing) {
	   PAINT( true , true , true , false , false )
	} else if (g_bUseSpecular && g_bUsePhongInterp && g_bUseReflections && !g_bUseShadows && g_bUseAntialiasing) {
	   PAINT( true , true , true , false , true )
	} else if (g_bUseSpecular && g_bUsePhongInterp && g_bUseReflections && g_bUseShadows && !g_bUseAntialiasing) {
	   PAINT( true , true , true , true , false )
	} else if (g_bUseSpecular && g_bUsePhongInterp && g_bUseReflections && g_bUseShadows && g_bUseAntialiasing) {
	   PAINT( true , true , true , true , true )
	}
    }
    hipError_t error = hipGetLastError();
    if(error != hipSuccess) {
	printf("CUDA error: %s\n", hipGetErrorString(error));
	exit(-1);
    }

    SAFE(hipDeviceSynchronize());
    SAFE(cudaGLUnmapBufferObject(buffer));

    // Use OpenGL texture to display the generated frame at lightning speed
    // (the PBO buffer is already on the card, no useless PCI bus criss-cross)
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, buffer);
    glTexSubImage2D(GL_TEXTURE_2D, 0, 0, 0, MAXX, MAXY, GL_RGBA, GL_UNSIGNED_BYTE, NULL);
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, 0);

    glEnable(GL_TEXTURE_2D);
    glBegin(GL_QUADS);
    glTexCoord2f(0.0f, 1.0f); glVertex2f(-1.0f, -1.0f);
    glTexCoord2f(1.0f, 1.0f); glVertex2f(1.0f, -1.0f);
    glTexCoord2f(1.0f, 0.0f); glVertex2f(1.0f, 1.0f);
    glTexCoord2f(0.0f, 0.0f); glVertex2f(-1.0f, 1.0f);
    glEnd();

    // hack:
    extern bool g_benchmark;
    if (!g_benchmark) {
	// Display the "Press H for help" message
	glDisable(GL_LIGHTING);
	glDisable(GL_TEXTURE_2D);
	glColor3f(1.f, 1.f, 1.f);
	glRasterPos2f(-0.95, 0.9);
	const char *help = "Press H for help";
	for(unsigned o=0;o<strlen(help); o++)
	    glutBitmapCharacter(GLUT_BITMAP_HELVETICA_12, help[o]);
    }
    SDL_GL_SwapBuffers();
}
